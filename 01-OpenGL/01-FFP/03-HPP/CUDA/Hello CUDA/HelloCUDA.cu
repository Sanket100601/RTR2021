// header files
// standard headers
#include <stdio.h>

// cuda headers
#include <hip/hip_runtime.h>

// global variable 
const int iNumberOfArrayElements = 5;

float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostOutput = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceOutput = NULL;

// Cuda kernel 
__global__ void vecAddGPU(float* in1, float* in2, float* out, int len)
{
	// code
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

// entry-point function
int main(void)
{
	//function declarations
	void cleanup(void);

	// variable declarations
	int size = iNumberOfArrayElements * sizeof(float);
	hipError_t result = hipSuccess;

	// code
	// host memeory allocation
	hostInput1 = (float*)malloc(size);
	if (hostInput1 == NULL)
	{ 
		printf("Host Memory allocation is failed for hostInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2 = (float*)malloc(size);
	if(hostInput2==NULL)
	{
		printf("Host Memory allocation if failed for hostInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostOutput = (float*)malloc(size);
	if (hostOutput == NULL)
	{ 
		printf("Host Memeory allocation is failed for hostOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// filling  values into host array
	hostInput1[0] = 101.0;
	hostInput1[1] = 102.0;
	hostInput1[2] = 103.0;
	hostInput1[3] = 104.0;
	hostInput1[4] = 105.0;

	hostInput2[0] = 201.0;
	hostInput2[1] = 202.0;
	hostInput2[2] = 203.0;
	hostInput2[3] = 204.0;
	hostInput2[4] = 205.0;

	// device memory allocation 
	result = hipMalloc((void**)&deviceInput1, size);
	if(result != hipSuccess)
	{
		printf("Device Memory allocation is failed for deviceInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void**)&deviceInput2, size);
	if(result != hipSuccess)
	{ 
		printf("Device Memory allocation is failed for deviceInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void**)&deviceOutput, size);
	if(result != hipSuccess)
	{ 
		printf("Device Memory allocation is failed for deviceOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// copy data form host arrays into device arrays 
	result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{ 
		printf("Host to Device Data Copy is failed for deviceInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{ 
		printf("Host to Device data copy is failed for deviceInput2 array .\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	dim3 dimGrid = dim3(iNumberOfArrayElements, 1, 1);
	dim3 dimBlock = dim3(1,1,1);

	// CUDA kernal for vector Addition 
	vecAddGPU <<<dimGrid, dimBlock >>> (deviceInput1, deviceInput2, deviceOutput, iNumberOfArrayElements);

	// copy data from device array into host array 
	result = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
	if (result != hipSuccess)
	{ 
		printf("Device to host data copy is failed for hostOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// vector addition on host 
	for (int i = 0; i < iNumberOfArrayElements; i++)
	{ 
		printf(" %f + %f = %f\n", hostInput1[i], hostInput2[i], hostOutput[i]);
	}

	// cleanup
	cleanup();

	return(0);
}

 void cleanup(void)
{ 
	// code 
	if (deviceOutput)
	{ 
		hipFree(deviceOutput);
		deviceOutput = NULL;
	}

	if (deviceInput2)
	{ 
		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}

	if (deviceInput1)
	{ 
		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}

	if (hostOutput)
	{
		free(hostOutput);
		hostOutput = NULL;
	}

	if (hostInput2)
	{
		free(hostInput2);
		hostInput2 = NULL;
	}

	if (hostInput1)
	{ 
		free(hostInput1);
		hostInput1 = NULL;

	}
}