// sinewave kernel

#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>
#include<stdio.h>
__global__ void sinewave(float4 *pos, unsigned int width, unsigned int height,
                         float animTime) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  float u = x / (float)width;
  float v = y / (float)height;

  u = (u * 2.0) - 1.0;
  v = (v * 2.0) - 1.0;

  float freq = 10.0f;
  float w = sinf(freq * u + animTime) * cosf(freq * v + animTime) * 0.02f;
  /*float freq = 10.0f;
  float w = 1 - abs(sinf(freq * u + time)) * abs(cosf(freq * v + time) * abs(cosf(freq * v + time)) * 0.4f);*/

  pos[y * width + x] = make_float4(u, w, v, 1.0f);
  return;
}

void launchCUDAKernel(float4 *pos, unsigned int width, unsigned int height,
                      float animTime) {
  dim3 block(8, 8, 1);
  dim3 grid(width / block.x, height / block.y, 1);

  sinewave<<<grid, block>>>(pos, width, height, animTime);
}