// sinewave kernel

#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>
#include<stdio.h>
__global__ void sinewave(float4 *pos, unsigned int width, unsigned int height,
                         float animTime) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  float u = x / (float)width;
  float v = y / (float)height;

  u = (u * 2.0) - 1.0;
  v = (v * 2.0) - 1.0;

 /* float freq = 10.0f;
  float w = sinf(freq * u + animTime) * cosf(freq * v + animTime) * 0.02f;*/
  float t = 0.01 * (-animTime * 130.0);
  float freq = 6.0f;
  float amp = 1.0f;

  float w = (sinf(freq * (u * v) + t)) * 0.1  /** sinf(freq * v + t)*/;
  w += (sinf((freq * 2.1 * (u * v)) + t)) * 0.492f;
  w += (sinf((freq * 1.72 * (u * v)) + (t * 1.121))) * 0.4f;
  w += (sinf((freq * 2.221 * (u * v)) + (t * 0.437))) * 0.5f;
  w += (sinf((freq * 3.1122 * (u * v)) + (t * 4.269))) * 0.25f;
  w *= amp * 0.3;

  pos[y * width + x] = make_float4(u, w, v, 1.0f);
  return;
}

void launchCUDAKernel(float4 *pos, unsigned int width, unsigned int height,
                      float animTime) {
  dim3 block(8, 8, 1);
  dim3 grid(width / block.x, height / block.y, 1);

  sinewave<<<grid, block>>>(pos, width, height, animTime);
}